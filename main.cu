#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel() {
    std::printf("Hello world!\n");
}

int main() {
    kernel<<<1,1>>>();
    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
    }
    return 0;
}
